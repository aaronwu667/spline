#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <time.h>

/* To-do: put this in a header file */
struct spline{
  int grid_size;
  int nqty;
  float *x;
  float *fx;
  float *coeff;
};


// Error checking macro due to Wenjie
#define cudaCheckErrors(msg)			\
  do {						\
    hipError_t __err = hipGetLastError();	\
    if (__err != hipSuccess) {				       \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",       \
	      msg, hipGetErrorString(__err),		       \
	      __FILE__, __LINE__);			       \
      fprintf(stderr, "*** FAILED - ABORTING\n");	       \
      exit(1);						       \
    }							       \
  } while(0)

__global__ void spline_knot_system(struct spline *spl, int grid_idx, int grid_size){
  int nqty_idx = blockIdx.x;
  int n = grid_size;
  if(grid_idx == 0){
    spl->coeff[0] = 3*(((spl->fx[(n * nqty_idx) + 2]-spl->fx[(n*nqty_idx) + 1])/
			(spl->x[2]-spl->x[1]))-
		       ((spl->fx[(n*nqty_idx)+1]-spl->fx[(n*nqty_idx)])/
			(spl->x[1]-spl->x[0])));
  }
  else if(grid_idx == n - 1){
    spl->coeff[n-1] = 3*(((spl->fx[(n*nqty_idx)+n-1]-spl->fx[(n*nqty_idx)+n-2])/
			  (spl->x[n-1]-spl->x[n-2]))-
			 ((spl->fx[(n*nqty_idx)+n-2]-spl->fx[(n*nqty_idx)+n-3])/
			  (spl->x[n-2]-spl->x[n-3])));
  }
  else{
    spl->coeff[grid_idx] = 3*(((spl->fx[(n*nqty_idx)+grid_idx+1]-spl->fx[(n*nqty_idx)+grid_idx])/(spl->x[grid_idx+1]-spl->x[grid_idx]))-
			      ((spl->fx[(n*nqty_idx)+grid_idx]-spl->fx[(n*nqty_idx)+grid_idx-1])/(spl->x[grid_idx]-spl->x[grid_idx-1])));
  }
}

__global__ void spline_knot_kernel(struct spline *spl, float* subdiag,
				   float* diag, float* superdiag){
  int grid_size = spl->grid_size;
  int grid_idx = blockIdx.x;
  if(grid_idx == 0){
    subdiag[0] = 0;
    
    diag[0] = (3 * (spl->x[1] - spl->x[0])) + (2 * (spl->x[2]-spl->x[1])) +
      (((spl->x[1]-spl->x[0]) * (spl->x[1]-spl->x[0]))/(spl->x[2]-spl->x[1]));
    
    superdiag[0] = (spl->x[2]-spl->x[1]) -
      (((spl->x[1]-spl->x[0]) * (spl->x[1]-spl->x[0]))/(spl->x[2]-spl->x[1]));
  }
  else if(grid_idx == grid_size-1){
    superdiag[grid_size-1] = 0;
    
    diag[grid_size - 1] = (3 * (spl->x[grid_size-1] - spl->x[grid_size-2])) +
      (2 * (spl->x[grid_size-2]-spl->x[grid_size-3])) +
    (((spl->x[grid_size-1]-spl->x[grid_size-2]) *
      (spl->x[grid_size-1]-spl->x[grid_size-2]))/
     (spl->x[grid_size-2]-spl->x[grid_size-3]));
    
    subdiag[grid_size - 1] = (spl->x[grid_size-2]-spl->x[grid_size-3]) -
      (((spl->x[grid_size-1]-spl->x[grid_size-2]) *
      (spl->x[grid_size-1]-spl->x[grid_size-2]))/
     (spl->x[grid_size-2]-spl->x[grid_size-3]));
  }
  else{
    diag[grid_idx] =  2 * ((spl->x[grid_idx] - spl->x[grid_idx-1]) +
			   (spl->x[grid_idx+1] - spl->x[grid_idx]));
    subdiag[grid_idx] = spl->x[grid_idx]-spl->x[grid_idx-1];
    superdiag[grid_idx]= spl->x[grid_idx+1]-spl->x[grid_idx];
  }
  
  spline_knot_system<<<spl->nqty,1>>>(spl, grid_idx, grid_size);
}


__global__ void eval_kernel_nqty(struct spline *spl, float* values, float* eval_pts,
				 int pt_idx, int floor_idx, int n, int numEval){

  int nqty_idx = blockIdx.x;
  float b_i = ((spl->fx[(n*nqty_idx)+floor_idx+1] - spl->fx[(n*nqty_idx)+floor_idx])/(spl->x[floor_idx+1] - spl->x[floor_idx])) -
    (((spl->x[floor_idx+1] - spl->x[floor_idx])*(spl->coeff[(n*nqty_idx)+floor_idx+1]+(2*spl->coeff[(n*nqty_idx)+floor_idx])))/3);
  
  float d_i = (spl->coeff[(n*nqty_idx)+floor_idx+1] - spl->coeff[(n*nqty_idx)+floor_idx])/(3*(spl->x[floor_idx+1] - spl->x[floor_idx]));
  
  values[(numEval*nqty_idx) + pt_idx] = spl->fx[(n*nqty_idx)+floor_idx] + (b_i*(eval_pts[pt_idx]-spl->x[floor_idx])) +
    (spl->coeff[(n*nqty_idx) + floor_idx]*((eval_pts[pt_idx]-spl->x[floor_idx])*(eval_pts[pt_idx]-spl->x[floor_idx]))) +
    (d_i*((eval_pts[pt_idx]-spl->x[floor_idx])*(eval_pts[pt_idx]-spl->x[floor_idx])*
	  (eval_pts[pt_idx]-spl->x[floor_idx]))); 
}



__global__ void spline_eval_kernel(struct spline *spl, float* values, float* eval_pts, int numEval){
  int pt_idx = blockIdx.x;
  int floor_idx;
  int n = spl->grid_size;
  /* linear search for index */
  if(eval_pts[pt_idx] <= spl->x[0]){
    floor_idx = 0;
  }
  else if(eval_pts[pt_idx] >= spl->x[n-1]){
    floor_idx = n-2;
  }
  for(int i = 0; i<n; i++){
    if(eval_pts[pt_idx]<spl->x[i]){
      floor_idx = i-1;
      break;
    }
  }
  eval_kernel_nqty<<<spl->nqty,1>>>(spl, values, eval_pts, pt_idx, floor_idx, n, numEval);
}


void spline_knot_fit(struct spline *spl, float* coeff, float* scratch, void* buffer, int grid_size, int nqty){


  float* subdiag = scratch;
  float* diag = subdiag + grid_size;
  float* superdiag = diag + grid_size;
  
  spline_knot_kernel<<<grid_size,1>>>(spl, subdiag, diag, superdiag);

  
  hipsparseStatus_t status;
  hipsparseHandle_t handle = 0;
  status = hipsparseCreate(&handle);
  if(status != HIPSPARSE_STATUS_SUCCESS){
    fprintf(stderr, "cuSparse failed to initialize");
  }
  
  
 
  hipsparseSgtsv2_nopivot(handle, grid_size, nqty, subdiag,
			 diag, superdiag, coeff, grid_size, buffer);
 
  
}

void spline_eval(struct spline *spl, float* values, float* eval_pts,
		 int numEval, float* scratch, int fitFlag){
  /*
  if(fitFlag){
    spline_knot_fit(spl, scratch);
  }
  */
  spline_eval_kernel<<<numEval,1>>>(spl, values, eval_pts, numEval);
}




int main(){
  float x[500];
  float y[1000000];
  int n = 500;
  float c[1000000];
  float scratch[5000];

   
  FILE* fx = fopen("fx.dat", "r");
  FILE* x_grid = fopen("x_grid.dat", "r");
  for(int i = 0; i<1000000; i++){
    float entry;
    fscanf(fx,"%f", &entry);
    y[i] = entry; 
  }

  for(int i =0; i<500; i++){
    float entry;
    fscanf(x_grid, "%f", &entry);
    x[i] = entry;
  }
    

  
  struct spline test;
  test.grid_size = 500;
  test.nqty = 2000;
  
  struct spline *dtest;
  float* dx;
  float* dy;
  float* cd;
  float* dscratch;

  
  hipMalloc((void **)&dtest, sizeof(struct spline));
  hipMalloc((void**)&cd, sizeof(float)*1000000);
  hipMalloc((void**)&dscratch, sizeof(float) * 5000);
  hipMalloc((void **)&dx, sizeof(float) * 500);
  hipMalloc((void **)&dy, sizeof(float) * 1000000);
  
  hipMemcpy(dx, x, sizeof(float) * 500,hipMemcpyHostToDevice);
  hipMemcpy(dy, y, sizeof(float) * 1000000,hipMemcpyHostToDevice);
 
 
  test.x = dx;
  test.fx = dy;
  test.coeff = cd;
  hipMemcpy(dtest, &test, sizeof(struct spline), hipMemcpyHostToDevice);
  void* buffer;
  hipMalloc(&buffer, test.grid_size*(3+test.nqty)*sizeof(float));
  
  clock_t begin = clock();
  spline_knot_fit(dtest, cd, dscratch, buffer, test.grid_size,test.nqty);

  clock_t end = clock();
  float time_spent = (float)(end - begin) / CLOCKS_PER_SEC;

  
  printf("Time: %f", time_spent);
  
  return 0;
}
